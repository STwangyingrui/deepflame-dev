#include "hip/hip_runtime.h"
#include "dfTestEqn.H"

// kernel functions

__global__ void warmup(int num_cells)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_cells)
        return;
}

__global__ void ldu_to_csr(int num_cells, int num_surfaces,
        const int *permute_index, const int *csr_row_index, const int *csr_diag_index,
        const double *lower, const double *upper, const double *diag,
        const double *A_csr_input, double *A_csr_output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_cells)
        return;

    // A_csr has one more element in each row: itself
    int row_index = csr_row_index[index];
    int next_row_index = csr_row_index[index + 1];
    int diag_index = csr_diag_index[index];
    int neighbor_offset = csr_row_index[index] - index;

    for (int i = row_index; i < next_row_index; i++)
    {
        int inner_index = i - row_index;
        // lower
        if (inner_index < diag_index)
        {
            int neighbor_index = neighbor_offset + inner_index;

            int perm_index = permute_index[neighbor_index];
            double value = lower[perm_index];
            A_csr_output[i] = A_csr_input[i] + value;
        }
        // diag
        if (inner_index == diag_index)
        {
            A_csr_output[i] = A_csr_input[i] + diag[index];
        }
        // upper
        if (inner_index > diag_index)
        {
            // upper, index - 1, consider of diag
            int neighbor_index = neighbor_offset + inner_index - 1;

            int perm_index = permute_index[neighbor_index];
            double value = upper[perm_index - num_surfaces];
            A_csr_output[i] = A_csr_input[i] + value;
        }
    }
}

__global__ void test_fvm_div_internal(int num_cells, int num_surfaces,
                                 const int *lower_index, const int *upper_index,
                                 const double *weight, const double *phi,
                                 double *lower, double *upper, double *diag)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_surfaces)
        return;

    double w = weight[index];
    double f = phi[index];

    lower[index] += (-w) * f;
    upper[index] += (1 - w) * f;

    int l = lower_index[index];
    int u = upper_index[index];
    atomicAdd(&(diag[l]), w * f);
    atomicAdd(&(diag[u]), (w - 1) * f);
}

// constructor
dfTestEqn::dfTestEqn(dfMatrixDataBase &dataBase, const std::string &modeStr, const std::string &cfgFile)
    : dataBase_(dataBase)
{
    stream = dataBase_.stream;

    num_cells = dataBase_.num_cells;
    num_faces = dataBase_.num_faces;
    num_surfaces = dataBase_.num_surfaces;
    cell_vec_bytes = dataBase_.cell_vec_bytes;
    csr_value_bytes = dataBase_.csr_value_bytes;
    csr_value_vec_bytes = dataBase_.csr_value_vec_bytes;

    d_A_csr_row_index = dataBase_.d_A_csr_row_index;
    d_A_csr_diag_index = dataBase_.d_A_csr_diag_index;
    d_A_csr_col_index = dataBase_.d_A_csr_col_index;

    checkCudaErrors(hipMalloc((void **)&d_A_csr, csr_value_vec_bytes));
    checkCudaErrors(hipMalloc((void **)&d_b, cell_vec_bytes));
}

void dfTestEqn::initializeTimeStep(const double *phi)
{
    // initialize matrix value
    checkCudaErrors(hipMemsetAsync(d_A_csr, 0, csr_value_vec_bytes, stream));
    checkCudaErrors(hipMemsetAsync(d_b, 0, cell_vec_bytes, stream));
    checkCudaErrors(hipMemsetAsync(dataBase_.d_lower, 0, num_surfaces * sizeof(double), stream));
    checkCudaErrors(hipMemsetAsync(dataBase_.d_upper, 0, num_surfaces * sizeof(double), stream));
    checkCudaErrors(hipMemsetAsync(dataBase_.d_diag, 0, num_cells * sizeof(double), stream));

    memcpy(dataBase_.h_phi_init, phi, num_surfaces * sizeof(double));
    checkCudaErrors(hipMemcpyAsync(dataBase_.d_try_phi, dataBase_.h_phi_init, num_surfaces * sizeof(double), hipMemcpyHostToDevice, stream));
}

void dfTestEqn::fvm_div()
{
    size_t threads_per_block = 1024;
    size_t blocks_per_grid = 1;

    // warmup
    fprintf(stderr, "warmup...\n");
    blocks_per_grid = (num_cells + threads_per_block - 1) / threads_per_block;
    warmup<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells);

    //使用event计算时间
    float time_elapsed=0;
    hipEvent_t start,stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    checkCudaErrors(hipEventRecord(start,0));
    blocks_per_grid = (num_surfaces + threads_per_block - 1) / threads_per_block;
    test_fvm_div_internal<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells, num_surfaces,
            dataBase_.d_lowerAddr, dataBase_.d_upperAddr,
            dataBase_.d_try_weight, dataBase_.d_try_phi, dataBase_.d_lower, dataBase_.d_upper, dataBase_.d_diag);
/*
    // 
    blocks_per_grid = (num_cells + threads_per_block - 1) / threads_per_block;
    ldu_to_csr<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells, num_surfaces,
            dataBase_.d_permedIndex, d_A_csr_row_index, d_A_csr_diag_index,
            dataBase_.d_lower, dataBase_.d_upper, dataBase_.d_diag, d_A_csr, d_A_csr);
    checkCudaErrors(hipMemcpy(&d_A_csr[(num_cell + num_face)], &d_A_csr[0], csr_value_bytes, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(&d_A_csr[(num_cell + num_face) * 2], &d_A_csr[0], csr_value_bytes, hipMemcpyDeviceToDevice));
*/
    checkCudaErrors(hipEventRecord(stop,0));
    checkCudaErrors(hipEventSynchronize(start));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&time_elapsed,start,stop));
    fprintf(stderr, "try fvm_div_internal 执行时间：%f(ms)\n",time_elapsed);
}

void dfTestEqn::checkResult(const double *lower, const double *upper, const double *diag, bool print)
{
    std::vector<double> h_lower(num_surfaces);
    checkCudaErrors(hipMemcpy(h_lower.data(), dataBase_.d_lower, num_surfaces * sizeof(double), hipMemcpyDeviceToHost));
    std::vector<double> h_upper(num_surfaces);
    checkCudaErrors(hipMemcpy(h_upper.data(), dataBase_.d_upper, num_surfaces * sizeof(double), hipMemcpyDeviceToHost));
    std::vector<double> h_diag(num_cells);
    checkCudaErrors(hipMemcpy(h_diag.data(), dataBase_.d_diag, num_cells * sizeof(double), hipMemcpyDeviceToHost));
    if (print) {
        for (int i = 0; i < num_surfaces; i++) {
            fprintf(stderr, "cpu lower[%d]: %.10lf, gpu lower[%d]: %.10lf\n", i, lower[i], i, h_lower[i]);
        }
        for (int i = 0; i < num_surfaces; i++) {
            fprintf(stderr, "cpu upper[%d]: %.10lf, gpu upper[%d]: %.10lf\n", i, upper[i], i, h_upper[i]);
        }
        for (int i = 0; i < num_cells; i++) {
            fprintf(stderr, "cpu diag[%d]: %.10lf, gpu diag[%d]: %.10lf\n", i, diag[i], i, h_diag[i]);
        }
    }
    checkVectorEqual(num_surfaces, lower, h_lower.data(), 1e-5);
    checkVectorEqual(num_surfaces, upper, h_upper.data(), 1e-5);
    checkVectorEqual(num_cells, diag, h_diag.data(), 1e-5);
}

dfTestEqn::~dfTestEqn()
{
}
